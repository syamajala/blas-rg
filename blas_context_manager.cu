#include "blas_context_manager.h"

hipblasHandle_t get_blas_handle()
{
  static __thread hipblasHandle_t handle;

  if(handle == NULL)
  {
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
      printf("CUBLAS initialization failed! Status: %d\n", stat);
    }
  }
  return handle;
}
