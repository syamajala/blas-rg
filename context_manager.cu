#include "context_manager.h"

hipsolverHandle_t get_handle()
{
  static __thread hipsolverHandle_t handle;

  if(handle == NULL)
  {
    hipsolverStatus_t stat;
    stat = hipsolverDnCreate(&handle);

    if (stat != HIPSOLVER_STATUS_SUCCESS)
    {
      printf("CUSOLVER initialization failed! Status: %d\n", stat);
    }
  }
  return handle;
}
