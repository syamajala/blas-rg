#include "context_manager.h"

void create_handle(hipblasHandle_t *handle)
{
  hipblasStatus_t stat;
  stat = hipblasCreate(handle);
  if (stat != HIPBLAS_STATUS_SUCCESS)
  {
    printf("CUBLAS initialization failed!\n");
  }
}
